
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

const int n = 1000;

__global__ void add(int *array, int n)
{
    int base = blockIdx.x * threadIdx.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;

    for(int i = base; i < offset;i++) {
        array[i] = array[i] * array[i];
    }
}

int main() {
    int *a = (int*)malloc(sizeof(int) * n);
    for(int i = 0; i < n;i++) a[i] = (int)i;

    int *cuda_buffer;
    hipMalloc((void**)&cuda_buffer, sizeof(int) * n);

    hipMemcpy(cuda_buffer, a, n * sizeof(int), hipMemcpyHostToDevice);

    add<<<1,128>>>(cuda_buffer, n);

    hipMemcpy(cuda_buffer, a, sizeof(int)*n, hipMemcpyDeviceToHost);

    for(int i = 0; i < n;i++) cout << a[i] << endl;

    hipFree(cuda_buffer);
    free(a);

    return 0;
}
