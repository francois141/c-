
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void init(int *a, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = idx; i < n;i += stride) a[i] = 1.0;
}

__global__ void reduce(int *a, int n,int *result) {
    unsigned int size = blockDim.x/2;
    const unsigned int index = threadIdx.x;
    const unsigned int blockIndex = blockIdx.x * blockDim.x;

    __shared__ int array[128];
    a[index + blockIndex] = 1.0;
    array[index] = a[index + blockIndex];

    __syncthreads();

    while(size > 0) {
        // Reduce warp
        if(index < size) {
            array[index] += array[index + size];
        }
        size /= 2;
        __syncthreads();
    }

    if(index == 0) {
        // Atomic add
        atomicAdd(result, array[0]);
    }
}

int main() {

    const int n = 10000 * 128;
    int *a;

    int *result;

    hipMallocManaged(&result, sizeof(int));
    hipMalloc(&a, sizeof(int)*n);

    init<<<1,1024>>>(a,n);
    reduce<<<ceil(n / 256),256>>>(a, n, result);

    hipDeviceSynchronize();

    cout << *result << endl;

    return 0;
}