
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

const int n = 100;

__global__ void mult(float *a) {
    for(int i = 0; i <n;i++) a[i] = 2*a[i];
}

int main() {
    
    float *a = (float*)malloc(sizeof(float) * n);
    for(int i = 0; i < n;i++) a[i] = (float)i;

    float *a_d; 
    hipMalloc((void**)&a_d, sizeof(float) * n);

    cout << a[2] << endl;

    hipMemcpy(a_d, a, sizeof(float)*n, hipMemcpyHostToDevice);

    mult<<<1,1>>>(a_d);

    hipMemcpy(a, a_d, sizeof(float)*n, hipMemcpyDeviceToHost);

    hipFree(a_d);

    cout << a[2] << endl;
    free(a);

    return 0;
}
