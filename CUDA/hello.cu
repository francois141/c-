
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

const int n = 1000;

__global__ void find_maximum_kernel(int *array, int *max2, int *mutex, unsigned int n)
{
    unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
    unsigned int stride = gridDim.x*blockDim.x;

    const unsigned int SHARED_SIZE = 1024;
    __shared__ float cache[SHARED_SIZE];
    for (int i = threadIdx.x; i < SHARED_SIZE; i += blockDim.x) {
        cache[i] = 0;
    }

    __syncthreads();

    unsigned int offset = index;
    int temp = -1.0;

    while(offset < n){
        temp = max(temp, array[offset]);
        offset += stride;
    }

    cache[threadIdx.x] = temp;

    __syncthreads();

    // reduction
    // TODO: Write better code
    for(int i = 0; i < SHARED_SIZE;i++) {
        cache[0] = max(cache[0], cache[i]);
    }

    __syncthreads();

    if(threadIdx.x == 0){
        atomicMax(max2, cache[0]);
    }
}

int main() {
    
    int *a = (int*)malloc(sizeof(int) * n);
    for(int i = 0; i < n;i++) a[i] = (int)i;

    int *a_d;
    int *device_max;
    int host_max;
    int *mutex;

    hipMalloc((void**)&a_d, sizeof(int) * n);
    hipMalloc((void**)&mutex,sizeof(int));
    hipMalloc((void**)&device_max,sizeof(int));


    hipMemcpy(a_d, a, sizeof(int)*n, hipMemcpyHostToDevice);

    find_maximum_kernel<<<5,128>>>(a_d, device_max, mutex,  n);

    hipMemcpy(&host_max, device_max, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(a_d);

    cout << host_max << endl;
    free(a);

    return 0;
}
