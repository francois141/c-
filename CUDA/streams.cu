
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void add(int *a, int size){

    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * blockDim.x;

    for(int i = start; i < size;i += stride) {
        atomicAdd(&a[i], 12);
    }
}

int main() {

    const int n= 10000;

    int *a;
    hipMallocManaged(&a, sizeof(int) * n);
    a[0] = 2;

    hipStream_t streams[2];

    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);

    add<<<1,128,0, streams[0]>>>(a, n);
    add<<<1,128,0, streams[1]>>>(a, n);

    hipDeviceSynchronize();

    hipStreamDestroy(streams[0]);
    hipStreamDestroy(streams[1]);

    cout << a[0] << endl;


    return 0;
}